#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include "cuda_rasterizer/adam.h"
#include <fstream>
#include <string>
#include <functional>

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::function<int*(size_t N)> resizeIntFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return t.contiguous().data_ptr<int>();
    };
    return lambda;
}

std::function<float*(size_t N)> resizeFloatFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return t.contiguous().data_ptr<float>();
    };
    return lambda;
}

std::tuple<int, int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& dc,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug,
	const torch::Tensor& pixel_weights) // 正常渲染时为 0 Tensor；计算高斯得分时为 加权后的 当前视角的 渲染图像与gt的L1 loss和 gt图像的归一化纹理边缘, (H,W)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHAFFELS, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  torch::Tensor sampleBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  std::function<char*(size_t)> sampleFunc = resizeFunctional(sampleBuffer);
  
  torch::Tensor xy_d = torch::zeros({P, 2}, means3D.options().dtype(torch::kFloat32));
  torch::Tensor depths_d = torch::zeros({P}, means3D.options().dtype(torch::kFloat32));
  torch::Tensor radii_d = torch::zeros({P}, means3D.options().dtype(torch::kInt32));

  torch::Tensor countBuffer = torch::empty({0}, int_opts);
  torch::Tensor offsetBuffer = torch::empty({0}, int_opts);
  torch::Tensor listBuffer = torch::empty({0}, int_opts);
  torch::Tensor listBufferRender = torch::empty({0}, float_opts);
  torch::Tensor listBufferDistance = torch::empty({0}, float_opts);
  std::function<int*(size_t)> listFunc = resizeIntFunctional(listBuffer);
  std::function<float*(size_t)> listFuncRender = resizeFloatFunctional(listBufferRender);
  std::function<float*(size_t)> listFuncDistance = resizeFloatFunctional(listBufferDistance);

  float* weight_ptr = nullptr;
  float* accum_weight_ptr = nullptr;
  int* reverse_count_ptr = nullptr;
  float* blending_weight_ptr = nullptr;
  float* dist_accum_ptr = nullptr;

  torch::Tensor accumWeightsBuffer = torch::empty({0}, float_opts);
  torch::Tensor reverseCount = torch::empty({0}, int_opts);
  torch::Tensor blendingWeights = torch::empty({0}, float_opts);
  torch::Tensor distAccum = torch::empty({0}, float_opts);

  if(pixel_weights.size(0) != 0)    // 正常渲染时为 0 Tensor；计算高斯得分时为 加权后的 当前视角的 渲染图像与gt的L1 loss和 gt图像的归一化纹理边缘, (H,W)
  {
	countBuffer = torch::full({H, W}, 0, int_opts);
	offsetBuffer = torch::full({H, W}, 0, int_opts);

	if(pixel_weights.size(0) > 1)
	{
		weight_ptr = pixel_weights.contiguous().data<float>();
		accumWeightsBuffer = torch::full({P}, 0, float_opts);
		accum_weight_ptr = accumWeightsBuffer.contiguous().data<float>();

		reverseCount = torch::full({P}, 0, int_opts);
		reverse_count_ptr = reverseCount.contiguous().data<int>();
		blendingWeights = torch::full({P}, 0, float_opts);
		blending_weight_ptr = blendingWeights.contiguous().data<float>();
		distAccum = torch::full({P}, 0, float_opts);
		dist_accum_ptr = distAccum.contiguous().data<float>();
	}
  }
  
  int rendered = 0;
  int num_buckets = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }

	  auto tup = CudaRasterizer::Rasterizer::forward(
		(float2 *)xy_d.contiguous().data_ptr<float>(),
		depths_d.contiguous().data_ptr<float>(),
		radii_d.contiguous().data_ptr<int>(),
	    geomFunc,
		binningFunc,
		imgFunc,
		sampleFunc,
		listFunc,
		listFuncRender,
		listFuncDistance,
		countBuffer.contiguous().data_ptr<int>(),
		offsetBuffer.contiguous().data_ptr<int>(),
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		dc.contiguous().data_ptr<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug,
		weight_ptr,
		accum_weight_ptr,
		reverse_count_ptr,
		blending_weight_ptr,
		dist_accum_ptr);

		rendered = std::get<0>(tup);
		num_buckets = std::get<1>(tup);
  }
  return std::make_tuple(rendered, num_buckets, out_color, radii, geomBuffer, binningBuffer, imgBuffer, sampleBuffer, offsetBuffer, listBuffer, listBufferRender, listBufferDistance, xy_d, depths_d, radii_d, accumWeightsBuffer, reverseCount, blendingWeights, distAccum);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& colors,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const torch::Tensor& dL_dout_color,
	const torch::Tensor& dc,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const int B,
	const torch::Tensor& sampleBuffer,
	const bool debug) 
{
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHAFFELS}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_ddc = torch::zeros({P, 1, 3}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  
  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, degree, M, R, B,
	  background.contiguous().data<float>(),
	  W, H, 
	  means3D.contiguous().data<float>(),
	  dc.contiguous().data<float>(),
	  sh.contiguous().data<float>(),
	  colors.contiguous().data<float>(),
	  scales.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  cov3D_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  projmatrix.contiguous().data<float>(),
	  campos.contiguous().data<float>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(sampleBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data<float>(),
	  dL_dmeans2D.contiguous().data<float>(),
	  dL_dconic.contiguous().data<float>(),  
	  dL_dopacity.contiguous().data<float>(),
	  dL_dcolors.contiguous().data<float>(),
	  dL_dmeans3D.contiguous().data<float>(),
	  dL_dcov3D.contiguous().data<float>(),
	  dL_ddc.contiguous().data<float>(),
	  dL_dsh.contiguous().data<float>(),
	  dL_dscales.contiguous().data<float>(),
	  dL_drotations.contiguous().data<float>(),
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dcov3D, dL_ddc, dL_dsh, dL_dscales, dL_drotations);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}

void adamUpdate(
	torch::Tensor &param,
	torch::Tensor &param_grad,
	torch::Tensor &exp_avg,
	torch::Tensor &exp_avg_sq,
	torch::Tensor &visible,
	const float lr,
	const float b1,
	const float b2,
	const float eps,
	const uint32_t N,
	const uint32_t M
){
	ADAM::adamUpdate(
		param.contiguous().data<float>(),
		param_grad.contiguous().data<float>(),
		exp_avg.contiguous().data<float>(),
		exp_avg_sq.contiguous().data<float>(),
		visible.contiguous().data<bool>(),
		lr,
		b1,
		b2,
		eps,
		N,
		M);
}